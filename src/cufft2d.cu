#include "common.hpp"
#include <hipfft/hipfft.h>

void compute_2d_fft_cufft(const std::vector<float>& h_data, int width, int height, 
                         std::vector<hipfftComplex>& h_result) {
    int complex_cols = width / 2 + 1;
    
    // Allocate device memory
    hipfftReal* d_data_in;
    hipfftComplex* d_data_out;
    CHECK_CUDA(hipMalloc((void**)&d_data_in, sizeof(hipfftReal) * width * height));
    CHECK_CUDA(hipMalloc((void**)&d_data_out, sizeof(hipfftComplex) * height * complex_cols));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Start timing
    CHECK_CUDA(hipEventRecord(start));

    // Copy input data to device
    CHECK_CUDA(hipMemcpy(d_data_in, h_data.data(), 
                         sizeof(hipfftReal) * width * height, 
                         hipMemcpyHostToDevice));

    // Create cuFFT plan
    hipfftHandle plan;
    if (hipfftPlan2d(&plan, height, width, HIPFFT_R2C) != HIPFFT_SUCCESS) {
        std::cerr << "CUFFT plan creation failed" << std::endl;
        hipFree(d_data_in);
        hipFree(d_data_out);
        return;
    }

    // Execute FFT
    if (hipfftExecR2C(plan, d_data_in, d_data_out) != HIPFFT_SUCCESS) {
        std::cerr << "CUFFT execution failed" << std::endl;
        hipfftDestroy(plan);
        hipFree(d_data_in);
        hipFree(d_data_out);
        return;
    }

    // Copy results back to host
    h_result.resize(height * complex_cols);
    CHECK_CUDA(hipMemcpy(h_result.data(), d_data_out,
                         sizeof(hipfftComplex) * height * complex_cols,
                         hipMemcpyDeviceToHost));

    // Stop timing
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Total execution time: " << milliseconds << " ms" << std::endl;

    // Cleanup
    hipfftDestroy(plan);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_data_in);
    hipFree(d_data_out);
}

int main(int argc, char** argv) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <input_file>" << std::endl;
        return 1;
    }

    // Get matrix dimensions
    int width, height;
    get_matrix_dimensions(argv[1], width, height);
    std::cout << "Processing matrix of size " << width << "x" << height << std::endl;

    // Read input data
    std::vector<float> h_data_in;
    if (!read_matrix_data(argv[1], h_data_in, width, height)) {
        return 1;
    }

    // Convert input to single precision real
    std::vector<float> h_data_real(width * height);
    for (int i = 0; i < width * height; ++i) {
        h_data_real[i] = h_data_in[i * 2] / 100.0f;  // Normalize input
    }

    // Perform FFT
    std::vector<hipfftComplex> h_result;
    compute_2d_fft_cufft(h_data_real, width, height, h_result);



    return 0;
}